#include "hip/hip_runtime.h"
#include "util/util.hpp"
#include "kinfu.hpp"
#include <opencv2/opencv.hpp>






////////////////////////////////////////////////////////////////////////////////
// Import Marvin

#define DATATYPE 1 // Marvin datatype
#include "marvin.hpp"

////////////////////////////////////////////////////////////////////////////////
// Global variables for Marvin
std::string model_idx = "4";
marvin::Net main_net("models/model" + model_idx + ".test.json");

// Init marvin net
void init_marvin() {
  main_net.Malloc(marvin::Testing);
  std::vector<std::string> models = marvin::getStringVector("models/PeriodNet.5." + model_idx + "_snapshot_25000.marvin");
  for (int m=0;m<models.size();++m)   
    main_net.loadWeights(models[m]);
//     // marvin::Net net("tools/marvin/model" + model_idx + ".test.json");
//     main_net.Malloc(marvin::Testing);
//     std::vector<std::string> models = marvin::getStringVector("tools/marvin/PeriodNet.1." + model_idx + ".60000.marvin");
//     for (int m=0;m<models.size();++m)   
//       main_net.loadWeights(models[m]);
}

////////////////////////////////////////////////////////////////////////////////

void show_object_pose(float* K, float* object_pose, cv::Mat& display_frame) {

  // Compute center of ground truth object in 3D camera coordinates
  float object_center_display_3D[3] = {0};
  for (int i = 0; i < 3; i++)
    object_center_display_3D[i] = object_pose[i * 4 + 3];

  // Compute axis endpoints of ground truth object pose in 3D camera coordinates
  float object_pose_display_3D[18] = {0};
  for (int i = 0; i < 3; i++) {
    object_pose_display_3D[0 * 6 + i * 2 + 0] = object_pose[0 * 4 + 3] - object_pose[0 * 4 + i] * 0.07f;
    object_pose_display_3D[0 * 6 + i * 2 + 1] = object_pose[0 * 4 + 3] + object_pose[0 * 4 + i] * 0.07f;
    object_pose_display_3D[1 * 6 + i * 2 + 0] = object_pose[1 * 4 + 3] - object_pose[1 * 4 + i] * 0.07f;
    object_pose_display_3D[1 * 6 + i * 2 + 1] = object_pose[1 * 4 + 3] + object_pose[1 * 4 + i] * 0.07f;
    object_pose_display_3D[2 * 6 + i * 2 + 0] = object_pose[2 * 4 + 3] - object_pose[2 * 4 + i] * 0.07f;
    object_pose_display_3D[2 * 6 + i * 2 + 1] = object_pose[2 * 4 + 3] + object_pose[2 * 4 + i] * 0.07f;
  }

  // Project endpoints of ground truth object pose axis from 3D to 2D
  float object_pose_display_2D[12] = {0};
  for (int i = 0; i < 6; i++) {
    object_pose_display_2D[0 * 6 + i] = (object_pose_display_3D[0 * 6 + i] * K[0]) / (object_pose_display_3D[2 * 6 + i]) + K[2];
    object_pose_display_2D[1 * 6 + i] = (object_pose_display_3D[1 * 6 + i] * K[4]) / (object_pose_display_3D[2 * 6 + i]) + K[5];
  }
  // for (int i = 0; i < 12; i++)
  //   std::cout << object_pose_display_2D[i] << std::endl;

  // Project center of ground truth object from 3D to 2D
  float object_center_display_2D[2] = {0};
  object_center_display_2D[0] = (object_center_display_3D[0] * K[0]) / (object_center_display_3D[2]) + K[2];
  object_center_display_2D[1] = (object_center_display_3D[1] * K[4]) / (object_center_display_3D[2]) + K[5];
  // for (int i = 0; i < 12; i++)
  //   std::cout << object_pose_display_2D[i] << std::endl;

  // Display ground truth object pose
  cv::line(display_frame, cv::Point(object_pose_display_2D[0], object_pose_display_2D[6]), cv::Point(object_pose_display_2D[1], object_pose_display_2D[7]), cv::Scalar(0, 0, 255), 2);
  cv::line(display_frame, cv::Point(object_pose_display_2D[2], object_pose_display_2D[8]), cv::Point(object_pose_display_2D[3], object_pose_display_2D[9]), cv::Scalar(0, 255, 0), 2);
  cv::line(display_frame, cv::Point(object_pose_display_2D[4], object_pose_display_2D[10]), cv::Point(object_pose_display_2D[5], object_pose_display_2D[11]), cv::Scalar(255, 0, 0), 2);
  cv::circle(display_frame, cv::Point(object_center_display_2D[0], object_center_display_2D[1]), 4, cv::Scalar(0, 255, 255), -1);
  cv::circle(display_frame, cv::Point(object_pose_display_2D[0], object_pose_display_2D[6]), 4, cv::Scalar(0, 0, 255), -1);
  cv::circle(display_frame, cv::Point(object_pose_display_2D[2], object_pose_display_2D[8]), 4, cv::Scalar(0, 255, 0), -1);
  cv::circle(display_frame, cv::Point(object_pose_display_2D[4], object_pose_display_2D[10]), 4, cv::Scalar(255, 0, 0), -1);
  // cv::namedWindow("Object Pose", CV_WINDOW_AUTOSIZE);
  // cv::imshow("Object Pose", display_frame);
}

////////////////////////////////////////////////////////////////////////////////

__global__
void gen_hypothesis_labels(int num_hypothesis, unsigned short* tmp_hypothesis_locations, char* tmp_hypothesis_labels, unsigned short* tmp_hypothesis_crop_2D, float* tmp_K, float tmp_vox_unit, int* tmp_vox_size, float* tmp_vox_range_cam, float* tmp_vox_tsdf) {

  // Check kernel index
  int hypothesis_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (hypothesis_idx >= num_hypothesis)
    return;

  float tsdf_surface_threshold = 0.2f;
  int cube_dim = 30;

  // Fetch hypothesis location
  int x = (int)tmp_hypothesis_locations[0 * num_hypothesis + hypothesis_idx];
  int y = (int)tmp_hypothesis_locations[1 * num_hypothesis + hypothesis_idx];
  int z = (int)tmp_hypothesis_locations[2 * num_hypothesis + hypothesis_idx];

  // Check cube occupancy
  int cube_occ = 0;
  for (int i = -15; i < 15; i++)
    for (int j = -15; j < 15; j++)
      for (int k = -15; k < 15; k++) {
        int vox_idx = (z + k) * tmp_vox_size[0] * tmp_vox_size[1] + (y + j) * tmp_vox_size[0] + (x + i);
        if (tmp_vox_tsdf[vox_idx] < tsdf_surface_threshold)
          cube_occ++;
      }

  // Skip near empty cubes
  if (cube_occ < cube_dim * cube_dim / 2)
    return;

  // Convert cube location from grid to camera coordinates
  float x_cam = ((float)x + 1) * tmp_vox_unit + tmp_vox_range_cam[0 * 2 + 0];
  float y_cam = ((float)y + 1) * tmp_vox_unit + tmp_vox_range_cam[1 * 2 + 0];
  float z_cam = ((float)z + 1) * tmp_vox_unit + tmp_vox_range_cam[2 * 2 + 0];

  // If cube 2D projection is not in image bounds, cube is invalid
  float cube_rad = ((float) cube_dim) * tmp_vox_unit / 2;
  float cube_front[12] = {(x_cam + cube_rad), (x_cam + cube_rad), (x_cam - cube_rad), (x_cam - cube_rad),
                          (y_cam + cube_rad), (y_cam - cube_rad), (y_cam - cube_rad), (y_cam + cube_rad),
                          (z_cam - cube_rad), (z_cam - cube_rad), (z_cam - cube_rad), (z_cam - cube_rad)
                         };
  float cube_front_2D[8] = {};
  for (int i = 0; i < 4; i++) {
    cube_front_2D[0 * 4 + i] = cube_front[0 * 4 + i] * tmp_K[0] / cube_front[2 * 4 + i] + tmp_K[2];
    cube_front_2D[1 * 4 + i] = cube_front[1 * 4 + i] * tmp_K[4] / cube_front[2 * 4 + i] + tmp_K[5];
  }
  for (int i = 0; i < 8; i++)
    cube_front_2D[i] = roundf(cube_front_2D[i]);
  if (fmin(fmin(cube_front_2D[0], cube_front_2D[1]), fmin(cube_front_2D[2], cube_front_2D[3])) < 0 ||
      fmax(fmax(cube_front_2D[0], cube_front_2D[1]), fmax(cube_front_2D[2], cube_front_2D[3])) >= 640 ||
      fmin(fmin(cube_front_2D[4], cube_front_2D[5]), fmin(cube_front_2D[6], cube_front_2D[7])) < 0 ||
      fmax(fmax(cube_front_2D[4], cube_front_2D[5]), fmax(cube_front_2D[6], cube_front_2D[7])) >= 480)
    return;

  tmp_hypothesis_labels[hypothesis_idx] = (char)1;

  // Save 2D patch of cube's 2D project to image
  tmp_hypothesis_crop_2D[0 * num_hypothesis + hypothesis_idx] = (unsigned short)roundf(cube_front_2D[2]);
  tmp_hypothesis_crop_2D[1 * num_hypothesis + hypothesis_idx] = (unsigned short)roundf(cube_front_2D[6]);
  tmp_hypothesis_crop_2D[2 * num_hypothesis + hypothesis_idx] = (unsigned short)roundf(cube_front_2D[1] - cube_front_2D[2]);
  tmp_hypothesis_crop_2D[3 * num_hypothesis + hypothesis_idx] = (unsigned short)roundf(cube_front_2D[4] - cube_front_2D[6]);
}

////////////////////////////////////////////////////////////////////////////////

void detect(const std::string &sequence_directory, const std::string &frame_prefix) {

  std::cout << "CPU: Loading RGB-D frame and camera info." << std::endl;

  // Load intrinsics (3x3 matrix)
  std::string intrinsic_filename = sequence_directory + "/intrinsics.K.txt";
  std::vector<float> K_vec = load_matrix_from_file(intrinsic_filename, 3, 3);
  float * K = &K_vec[0];
  // for (int i = 0; i < 9; i++)
  //   std::cout << K[i] << std::endl;

  // Load RGB-D frame
  std::string curr_frame_color_filename = sequence_directory + "/" + frame_prefix + ".color.png";
  cv::Mat curr_frame_color = cv::imread(curr_frame_color_filename.c_str(), 1);
  std::string curr_frame_depth_filename = sequence_directory + "/" + frame_prefix + ".depth.png";
  cv::Mat curr_frame_depth = cv::imread(curr_frame_depth_filename.c_str(), CV_LOAD_IMAGE_UNCHANGED);

  // Load image/depth/extrinsic data for current frame
  unsigned short * depth_data = (unsigned short *) malloc(480 * 640 * sizeof(unsigned short));
  for (int i = 0; i < 480 * 640; i++) {
    depth_data[i] = (((unsigned short) curr_frame_depth.data[i * 2 + 1]) << 8) + ((unsigned short) curr_frame_depth.data[i * 2 + 0]);
    // std::cout << depth_data[i] << std::endl;
  }

  std::cout << "GPU: Fusing depth into TSDF volume." << std::endl;
  // Compute relative camera pose transform between current frame and base frame
  // Compute camera view frustum bounds within the voxel volume
  float camera_relative_pose[16] = {0};
  float view_bounds[6] = {0};
  std::vector<float> curr_extrinsic;
  for (int i = 0; i < 3; i++) {
    curr_extrinsic.push_back(1.0f);
    for (int i = 0; i < 4; i++) {
      curr_extrinsic.push_back(0.0f);
    }
  }
  curr_extrinsic.push_back(1.0f);
  std::vector<std::vector<float>> extrinsics;
  extrinsics.push_back(curr_extrinsic);
  get_frustum_bounds(K, extrinsics, 0, 0, camera_relative_pose, view_bounds,
                     vox_unit, vox_size, vox_range_cam);

  // Copy fusion params to GPU
  kCheckCUDA(__LINE__, hipMemcpy(d_K, K, 9 * sizeof(float), hipMemcpyHostToDevice));
  kCheckCUDA(__LINE__, hipMemcpy(d_depth_data, depth_data, 480 * 640 * sizeof(unsigned short), hipMemcpyHostToDevice));
  kCheckCUDA(__LINE__, hipMemcpy(d_view_bounds, view_bounds, 6 * sizeof(float), hipMemcpyHostToDevice));
  kCheckCUDA(__LINE__, hipMemcpy(d_camera_relative_pose, camera_relative_pose, 16 * sizeof(float), hipMemcpyHostToDevice));
  kCheckCUDA(__LINE__, hipMemcpy(d_vox_range_cam, vox_range_cam, 6 * sizeof(float), hipMemcpyHostToDevice));

  // Integrate
  int num_blocks = vox_size[2];
  int num_threads = vox_size[1];
  integrate <<< num_blocks, num_threads >>>(d_K, d_depth_data, d_view_bounds, d_camera_relative_pose,
      vox_unit, vox_mu, d_vox_size, d_vox_range_cam, d_vox_tsdf, d_vox_weight);
  kCheckCUDA(__LINE__, hipGetLastError());
  // kCheckCUDA(__LINE__, hipDeviceSynchronize());

  // Copy data back to memory
  kCheckCUDA(__LINE__, hipMemcpy(vox_tsdf, d_vox_tsdf, vox_size[0] * vox_size[1] * vox_size[2] * sizeof(float), hipMemcpyDeviceToHost));
  kCheckCUDA(__LINE__, hipMemcpy(vox_weight, d_vox_weight, vox_size[0] * vox_size[1] * vox_size[2] * sizeof(float), hipMemcpyDeviceToHost));

  // Save curr volume to pointcloud file
  std::string scene_ply_name = "volume.pointcloud.ply";
  save_volume_to_ply(scene_ply_name, vox_size, vox_tsdf);

  // // Save curr volume to raw file
  // std::string volume_name = "volume.tsdf.bin";
  // std::ofstream outFile(volume_name, std::ios::binary | std::ios::out);
  // for (int i = 0; i < vox_size[0] * vox_size[1] * vox_size[2]; i++)
  //   outFile.write((char*)&vox_tsdf[i], sizeof(float));
  // outFile.close();

  std::cout << "GPU: Exhaustively generating sliding windows for object detection." << std::endl;

  // Compute bounding box of surface in TSDF volume
  float tsdf_surface_threshold = 0.2f;
  float grid_bounds[6] = {0};
  grid_bounds[0] = vox_size[0]; grid_bounds[2] = vox_size[1]; grid_bounds[4] = vox_size[2];
  for (int i = 0; i < vox_size[0] * vox_size[1] * vox_size[2]; i++) {
    if (std::abs(vox_tsdf[i]) < tsdf_surface_threshold) {
      float z = (float) (floor(i / (vox_size[0] * vox_size[1])));
      float y = (float) (floor((i - (z * vox_size[0] * vox_size[1])) / vox_size[0]));
      float x = (float) (i - (z * vox_size[0] * vox_size[1]) - (y * vox_size[0]));
      grid_bounds[0] = std::min(x, grid_bounds[0]); grid_bounds[1] = std::max(x, grid_bounds[1]);
      grid_bounds[2] = std::min(y, grid_bounds[2]); grid_bounds[3] = std::max(y, grid_bounds[3]);
      grid_bounds[4] = std::min(z, grid_bounds[4]); grid_bounds[5] = std::max(z, grid_bounds[5]);
    }
  }

  // Double check bounding box is not near edge of TSDF volume
  grid_bounds[0] = std::max(grid_bounds[0], 15.0f); grid_bounds[1] = std::min(grid_bounds[1], (float)vox_size[0] - 15.0f - 1.0f);
  grid_bounds[2] = std::max(grid_bounds[2], 15.0f); grid_bounds[3] = std::min(grid_bounds[3], (float)vox_size[1] - 15.0f - 1.0f);
  grid_bounds[4] = std::max(grid_bounds[4], 15.0f); grid_bounds[5] = std::min(grid_bounds[5], (float)vox_size[2] - 15.0f - 1.0f);
  // std::cout << grid_bounds[0] << " " << grid_bounds[1] << std::endl;
  // std::cout << grid_bounds[2] << " " << grid_bounds[3] << std::endl;
  // std::cout << grid_bounds[4] << " " << grid_bounds[5] << std::endl;
  int grid_size[3] = {0};
  int hop_size = 4;
  grid_size[0] = grid_bounds[1] - grid_bounds[0] + 1;
  grid_size[1] = grid_bounds[3] - grid_bounds[2] + 1;
  grid_size[2] = grid_bounds[5] - grid_bounds[4] + 1;
  for (int i = 0; i < 3; i++)
    grid_size[i] = (int)std::ceil(((float)grid_size[i])/((float)hop_size));

  // Create list of hypothesis cubes (store grid locations, and is valid or not (0 for invalid, 1 for positive, 2 for negative))
  int num_hypothesis = grid_size[0] * grid_size[1] * grid_size[2];
  unsigned short * hypothesis_locations = new unsigned short[3 * num_hypothesis];
  char * hypothesis_labels = new char[num_hypothesis];
  memset(hypothesis_labels, 0, sizeof(char) * num_hypothesis);
  for (int z = grid_bounds[4]; z <= grid_bounds[5]; z = z + hop_size)
    for (int y = grid_bounds[2]; y <= grid_bounds[3]; y = y + hop_size)
      for (int x = grid_bounds[0]; x <= grid_bounds[1]; x = x + hop_size) {
        int hypothesis_idx = (z - grid_bounds[4])/hop_size * grid_size[0] * grid_size[1] + 
                             (y - grid_bounds[2])/hop_size * grid_size[0] + 
                             (x - grid_bounds[0])/hop_size;
        // std::cout << x << " " << y << " " << z << std::endl;
        // std::cout << hypothesis_idx << std::endl;
        hypothesis_locations[0 * num_hypothesis + hypothesis_idx] = (unsigned short)x;
        hypothesis_locations[1 * num_hypothesis + hypothesis_idx] = (unsigned short)y;
        hypothesis_locations[2 * num_hypothesis + hypothesis_idx] = (unsigned short)z;
      }
  // std::cout << num_hypothesis << std::endl;
  // for (int i = 0; i < num_hypothesis; i++) {
  //   std::cout << hypothesis_locations[0 * num_hypothesis + i] << " " << hypothesis_locations[1 * num_hypothesis + i] << " " << hypothesis_locations[2 * num_hypothesis + i] << std::endl;
  // }

  // Copy list of hypothesis cubes to GPU memory
  unsigned short * d_hypothesis_locations;
  char * d_hypothesis_labels;
  kCheckCUDA(__LINE__, hipMalloc(&d_hypothesis_locations, 3 * num_hypothesis * sizeof(unsigned short)));
  kCheckCUDA(__LINE__, hipMalloc(&d_hypothesis_labels, num_hypothesis * sizeof(char)));
  kCheckCUDA(__LINE__, hipMemcpy(d_hypothesis_locations, hypothesis_locations, 3 * num_hypothesis * sizeof(unsigned short), hipMemcpyHostToDevice));
  kCheckCUDA(__LINE__, hipMemcpy(d_hypothesis_labels, hypothesis_labels, num_hypothesis * sizeof(char), hipMemcpyHostToDevice));

  // Copy hypothesis crop information to GPU memory
  unsigned short * d_hypothesis_crop_2D;
  kCheckCUDA(__LINE__, hipMalloc(&d_hypothesis_crop_2D, 4 * num_hypothesis * sizeof(unsigned short)));

  // Run kernel to get labels for hypotheses
  num_threads = 512;
  num_blocks = (int)ceil(((float)num_hypothesis) / ((float)num_threads));
  gen_hypothesis_labels<<<num_blocks,num_threads>>>(num_hypothesis, d_hypothesis_locations, d_hypothesis_labels, d_hypothesis_crop_2D, d_K, vox_unit, d_vox_size, d_vox_range_cam, d_vox_tsdf);
  kCheckCUDA(__LINE__, hipGetLastError());

  // Copy 2D crop information back to CPU
  unsigned short * hypothesis_crop_2D = new unsigned short[4 * num_hypothesis];
  kCheckCUDA(__LINE__, hipMemcpy(hypothesis_labels, d_hypothesis_labels, num_hypothesis * sizeof(char), hipMemcpyDeviceToHost));
  kCheckCUDA(__LINE__, hipMemcpy(hypothesis_crop_2D, d_hypothesis_crop_2D, 4 * num_hypothesis * sizeof(unsigned short), hipMemcpyDeviceToHost));

  int num_valid_hypotheses = 0;
  for (int i = 0; i < num_hypothesis; i++)
    if ((int)(hypothesis_labels[i]) == 1)
      num_valid_hypotheses++;

  // ROS_INFO("Found %d hypothesis bounding boxes.", num_valid_hypotheses);
  // ROS_INFO("Saving hypotheses to tensors on disk for Marvin.");
  std::cout << "GPU: Found " << num_valid_hypotheses << " hypothesis bounding boxes." << std::endl;
  std::cout << "CPU: Passing hypothesis bounding boxes to Marvin." << std::endl;

  buffer_data2D.clear();
  buffer_data3D.clear();
  buffer_data2D.resize(num_valid_hypotheses);
  buffer_data3D.resize(num_valid_hypotheses);

  // Write hypothesis cubes and patches to tensor file
  int valid_hypothesis_counter = 0;
  for (int hypothesis_idx = 0; hypothesis_idx < num_hypothesis; hypothesis_idx++) {
    if ((int)(hypothesis_labels[hypothesis_idx]) == 1) {
      int x = hypothesis_locations[0 * num_hypothesis + hypothesis_idx];
      int y = hypothesis_locations[1 * num_hypothesis + hypothesis_idx];
      int z = hypothesis_locations[2 * num_hypothesis + hypothesis_idx];

      // Get 3D cube
      float * curr_cube = new float[30 * 30 * 30];
      for (int i = -15; i < 15; i++)
        for (int j = -15; j < 15; j++)
          for (int k = -15; k < 15; k++) {
            int volumeIDX = (z + k) * vox_size[0] * vox_size[1] + (y + j) * vox_size[0] + (x + i);
            curr_cube[(k + 15) * 30 * 30 + (j + 15) * 30 + (i + 15)] = vox_tsdf[volumeIDX];
          }

      // Get 2D patch of cube's 2D project to image
      cv::Rect curr_patch_ROI(hypothesis_crop_2D[0 * num_hypothesis + hypothesis_idx], hypothesis_crop_2D[1 * num_hypothesis + hypothesis_idx], hypothesis_crop_2D[2 * num_hypothesis + hypothesis_idx], hypothesis_crop_2D[3 * num_hypothesis + hypothesis_idx]);
      // std::cout << std::round(cube_front_2D[2]) << " " << std::round(cube_front_2D[6]) << " " << std::round(cube_front_2D[1]-cube_front_2D[2]) << " " << std::round(cube_front_2D[4]-cube_front_2D[6]) << std::endl;
      cv::Mat curr_patch = curr_frame_color(curr_patch_ROI);
      cv::resize(curr_patch, curr_patch, cv::Size(227, 227));

      // Write 2D image patch to data tensor file (bgr and subtract mean)
      float * patch_data = new float[3 * 227 * 227];
      for (int tmp_row = 0; tmp_row < 227; tmp_row++)
        for (int tmp_col = 0; tmp_col < 227; tmp_col++) {
          patch_data[0 * 227 * 227 + tmp_row * 227 + tmp_col] = ((float) curr_patch.at<cv::Vec3b>(tmp_row, tmp_col)[0]) - 102.9801f; // B
          patch_data[1 * 227 * 227 + tmp_row * 227 + tmp_col] = ((float) curr_patch.at<cv::Vec3b>(tmp_row, tmp_col)[1]) - 115.9465f; // G
          patch_data[2 * 227 * 227 + tmp_row * 227 + tmp_col] = ((float) curr_patch.at<cv::Vec3b>(tmp_row, tmp_col)[2]) - 122.7717f; // R
        }
      buffer_data2D[valid_hypothesis_counter].resize(3 * 227 * 227);
      for (int i = 0; i < 3 * 227 * 227; i++)
        buffer_data2D[valid_hypothesis_counter][i] = patch_data[i];

      // Write 3D tsdf volume to data tensor file
      buffer_data3D[valid_hypothesis_counter].resize(30 * 30 * 30);
      for (int i = 0; i < 30 * 30 * 30; i++)
        buffer_data3D[valid_hypothesis_counter][i] = curr_cube[i];

      valid_hypothesis_counter++;

      // Clear memory
      delete [] patch_data;
      delete [] curr_cube;
    }
  }

  // Clear memory
  free(depth_data);
  hipFree(d_hypothesis_locations);
  hipFree(d_hypothesis_labels);
  hipFree(d_hypothesis_crop_2D);
  toc();

  // Temporarily clear TSDF volume on GPU
  // hipFree(d_vox_tsdf);
  // hipFree(d_vox_weight);

  // Run marvin
  std::cerr << "GPU: Running Marvin for 2D/3D deep learning." << std::endl;
  // std::string class_score_tensor_filename = "TMP.class.response.tensor";
  // std::string quaternion_score_tensor_filename = "TMP.quaternion.response.tensor";
  // std::string translation_score_tensor_filename = "TMP.quaternion.response.tensor";
  // std::string axis_score_tensor_filename = "TMP.axis_score_response.tensor";
  // std::string angle_score_tensor_filename = "TMP.angle_score_response.tensor";
  if (true) {
    int itersPerSave = 0;
    main_net.test(marvin::getStringVector("class_score,quat_pred,trans_pred"), marvin::getStringVector(""), itersPerSave);
    // sys_command("cd src/apc_vision/tools/marvin; export LD_LIBRARY_PATH=LD_LIBRARY_PATH:/usr/local/cuda/lib64:/usr/local/cudnn/v4rc/lib64; ./marvin test model.json PeriodNet.marvin class_score ../../../../" + class_score_tensor_filename);
  }
  std::cout << "CPU: Extracting object pose information from Marvin results." << std::endl;

  // Parse Marvin scores
  float * class_score_raw = &buffer_scores_class[0];
  float * quaternion_score_raw = &buffer_scores_quaternion[0];
  float * translation_score_raw = &buffer_scores_translation[0];

  // List objects
  std::vector<std::string> object_names;
  // object_names.push_back("book");
  // object_names.push_back("duck");
  object_names.push_back("expo");
  // object_names.push_back("frog");
  object_names.push_back("glue");
  // object_names.push_back("plugs");
  // object_names.push_back("spark");

  float * highest_class_scores = new float[object_names.size()];
  for (int i = 0; i < object_names.size(); i++)
    highest_class_scores[i] = 0;
  int * best_guess_IDX = new int[object_names.size()];
  float * best_guess_quaternion = new float[object_names.size() * 4];
  float * best_guess_translation = new float[object_names.size() * 3];
  int valid_hypothesis_idx = 0;
  for (int hypothesis_idx = 0; hypothesis_idx < num_hypothesis; hypothesis_idx++) {
    if ((int)(hypothesis_labels[hypothesis_idx]) == 1) {
      // std::cout << hypothesis_idx << " " << num_hypothesis << " " << valid_hypothesis_idx << std::endl;

      for (int j = 0; j < (object_names.size()+1); j++)
        std::cout << class_score_raw[valid_hypothesis_idx * (object_names.size()+1) + j] << " ";
      std::cout << std::endl;

      // Loop through each object type
      for (int i = 0; i < object_names.size(); i++) {

        // for (int j = 0; j < 4; j++)
        //   std::cout << std::to_string(i) << ": " << quaternion_score_raw[valid_hypothesis_idx * object_names.size() * 4 + i * 4 + j] << " ";
        // std::cout << std::endl;

        for (int j = 0; j < 3; j++)
          std::cout << std::to_string(i) << ": " << translation_score_raw[valid_hypothesis_idx * object_names.size() * 3 + i * 3 + j] << " ";
        std::cout << std::endl;

        float curr_class_score_raw = class_score_raw[valid_hypothesis_idx * (object_names.size() + 1) + i + 1];
        if (curr_class_score_raw > 0.5f) {

          // Draw top scoring boxes
          std::cout << valid_hypothesis_idx << std::endl;
          int crop_x1 = hypothesis_crop_2D[0 * num_hypothesis + hypothesis_idx];
          int crop_y1 = hypothesis_crop_2D[1 * num_hypothesis + hypothesis_idx];
          int crop_x2 = hypothesis_crop_2D[0 * num_hypothesis + hypothesis_idx] + hypothesis_crop_2D[2 * num_hypothesis + hypothesis_idx];
          int crop_y2 = hypothesis_crop_2D[1 * num_hypothesis + hypothesis_idx] + hypothesis_crop_2D[3 * num_hypothesis + hypothesis_idx];
          cv::rectangle(curr_frame_color, cv::Point(crop_x1, crop_y1), cv::Point(crop_x2, crop_y2), cv::Scalar(255, 0, 0));
          float object_location[3] = {0};
          object_location[0] = (hypothesis_locations[0 * num_hypothesis + hypothesis_idx] + 1) * vox_unit + vox_range_cam[0 * 2 + 0];
          object_location[1] = (hypothesis_locations[1 * num_hypothesis + hypothesis_idx] + 1) * vox_unit + vox_range_cam[1 * 2 + 0];
          object_location[2] = (hypothesis_locations[2 * num_hypothesis + hypothesis_idx] + 1) * vox_unit + vox_range_cam[2 * 2 + 0];
          float object_location_2D[2] = {0};
          object_location_2D[0] = (object_location[0] * K[0]) / (object_location[2]) + K[2];
          object_location_2D[1] = (object_location[1] * K[4]) / (object_location[2]) + K[5];
          cv::circle(curr_frame_color, cv::Point(object_location_2D[0], object_location_2D[1]), 4, cv::Scalar(255, 0, 0), -1);
          object_location[0] = object_location[0] + translation_score_raw[valid_hypothesis_idx * object_names.size() * 3 + i * 3 + 0]*(0.03f/std::sqrt(2.0f));
          object_location[1] = object_location[1] + translation_score_raw[valid_hypothesis_idx * object_names.size() * 3 + i * 3 + 1]*(0.03f/std::sqrt(2.0f));
          object_location[2] = object_location[2] + translation_score_raw[valid_hypothesis_idx * object_names.size() * 3 + i * 3 + 2]*(0.03f/std::sqrt(2.0f));
          object_location_2D[0] = (object_location[0] * K[0]) / (object_location[2]) + K[2];
          object_location_2D[1] = (object_location[1] * K[4]) / (object_location[2]) + K[5];
          cv::circle(curr_frame_color, cv::Point(object_location_2D[0], object_location_2D[1]), 4, cv::Scalar(0, 255, 0), -1);

          if (false) {
            float curr_quaternion[4] = {0};
            for (int j = 0; j < 4; j++)
              curr_quaternion[j] = quaternion_score_raw[valid_hypothesis_idx * object_names.size() * 4 + i * 4 + j];

            // Normalize quaternion 
            const float curr_quaternion_norm = 1.0f / sqrt(curr_quaternion[0] * curr_quaternion[0] + 
                                                                 curr_quaternion[1] * curr_quaternion[1] +
                                                                 curr_quaternion[2] * curr_quaternion[2] +
                                                                 curr_quaternion[3] * curr_quaternion[3]);
            curr_quaternion[0] *= curr_quaternion_norm;
            curr_quaternion[1] *= curr_quaternion_norm;
            curr_quaternion[2] *= curr_quaternion_norm;
            curr_quaternion[3] *= curr_quaternion_norm;
            std::cout << "Quaternion (normalized): " << curr_quaternion[0] << " " << curr_quaternion[1] << " " << curr_quaternion[2] << " " << curr_quaternion[3] << std::endl;

            // Convert quaternion to pose
            float object_pose[16];
            object_pose[0 * 4 + 0] = 1.0f - 2.0f * curr_quaternion[2] * curr_quaternion[2] - 2.0f * curr_quaternion[3] * curr_quaternion[3];
            object_pose[0 * 4 + 1] = 2.0f * curr_quaternion[1] * curr_quaternion[2] - 2.0f * curr_quaternion[3] * curr_quaternion[0];
            object_pose[0 * 4 + 2] = 2.0f * curr_quaternion[1] * curr_quaternion[3] + 2.0f * curr_quaternion[2] * curr_quaternion[0];
            object_pose[0 * 4 + 3] = object_location[0];
            object_pose[1 * 4 + 0] = 2.0f * curr_quaternion[1] * curr_quaternion[2] + 2.0f * curr_quaternion[3] * curr_quaternion[0];
            object_pose[1 * 4 + 1] = 1.0f - 2.0f * curr_quaternion[1] * curr_quaternion[1] - 2.0f * curr_quaternion[3] * curr_quaternion[3];
            object_pose[1 * 4 + 2] = 2.0f * curr_quaternion[2] * curr_quaternion[3] - 2.0f * curr_quaternion[1] * curr_quaternion[0];
            object_pose[1 * 4 + 3] = object_location[1];
            object_pose[2 * 4 + 0] = 2.0f * curr_quaternion[1] * curr_quaternion[3] - 2.0f * curr_quaternion[2] * curr_quaternion[0];
            object_pose[2 * 4 + 1] = 2.0f * curr_quaternion[2] * curr_quaternion[3] + 2.0f * curr_quaternion[1] * curr_quaternion[0];
            object_pose[2 * 4 + 2] = 1.0f - 2.0f * curr_quaternion[1] * curr_quaternion[1] - 2.0f * curr_quaternion[2] * curr_quaternion[2];
            object_pose[2 * 4 + 3] = object_location[2];
            object_pose[3 * 4 + 0] = 0.0f;
            object_pose[3 * 4 + 1] = 0.0f;
            object_pose[3 * 4 + 2] = 0.0f;
            object_pose[3 * 4 + 3] = 1.0f;

            // Display object pose
            show_object_pose(K, object_pose, curr_frame_color);
          }

          if (curr_class_score_raw > highest_class_scores[i]) {
            highest_class_scores[i] = curr_class_score_raw;
            best_guess_IDX[i] = hypothesis_idx;

            // Get best guess quaternion
            for (int j = 0; j < 4; j++)
              best_guess_quaternion[i * 4 + j] = quaternion_score_raw[valid_hypothesis_idx * object_names.size() * 4 + i * 4 + j];

            // Get best guess translation
            for (int j = 0; j < 3; j++)
              best_guess_translation[i * 3 + j] = translation_score_raw[valid_hypothesis_idx * object_names.size() * 3 + i * 3 + j];

          }
        }
      }
      valid_hypothesis_idx++;
    }
  }

      // std::cout << "got here" << std::endl;
  // // If no objects are detected
  // if (highest_class_scores[0] == 0 && highest_class_scores[1] == 0) {
  //   std::cout << "No objects detected!" << std::endl;
  //   object_names.clear();
  // }

  for (int object_idx = 0; object_idx < object_names.size(); object_idx++) {

    if (highest_class_scores[object_idx] == 0)
      continue;

    // Display detection results
    int crop_x1 = hypothesis_crop_2D[0 * num_hypothesis + best_guess_IDX[object_idx]];
    int crop_y1 = hypothesis_crop_2D[1 * num_hypothesis + best_guess_IDX[object_idx]];
    int crop_x2 = hypothesis_crop_2D[0 * num_hypothesis + best_guess_IDX[object_idx]] + hypothesis_crop_2D[2 * num_hypothesis + best_guess_IDX[object_idx]];
    int crop_y2 = hypothesis_crop_2D[1 * num_hypothesis + best_guess_IDX[object_idx]] + hypothesis_crop_2D[3 * num_hypothesis + best_guess_IDX[object_idx]];
    cv::rectangle(curr_frame_color, cv::Point(crop_x1, crop_y1), cv::Point(crop_x2, crop_y2), cv::Scalar(0, 255, 0), 2);
    // cv::circle(curr_frame_color, cv::Point((crop_x1 + crop_x2) / 2, (crop_y1 + crop_y2) / 2), 5, cv::Scalar(0, 255, 0), -1);

    std::cout << "Quaternion (raw): " << best_guess_quaternion[object_idx * 4 + 0] << " " << best_guess_quaternion[object_idx * 4 + 1] << " " << best_guess_quaternion[object_idx * 4 + 2] << " " << best_guess_quaternion[object_idx * 4 + 3] << std::endl;

    // Retrieve object location
    float object_location[3] = {0};
    object_location[0] = (hypothesis_locations[0 * num_hypothesis + best_guess_IDX[object_idx]] + 1) * vox_unit + vox_range_cam[0 * 2 + 0];
    object_location[1] = (hypothesis_locations[1 * num_hypothesis + best_guess_IDX[object_idx]] + 1) * vox_unit + vox_range_cam[1 * 2 + 0];
    object_location[2] = (hypothesis_locations[2 * num_hypothesis + best_guess_IDX[object_idx]] + 1) * vox_unit + vox_range_cam[2 * 2 + 0];

    // Apply translation prediction to object location
    std::cout << "Translation (raw): " << best_guess_translation[object_idx * 3 + 0] << " " << best_guess_translation[object_idx * 3 + 1] << " " << best_guess_translation[object_idx * 3 + 2] << std::endl;
    object_location[0] = object_location[0] + best_guess_translation[object_idx * 3 + 0]*(0.03f/std::sqrt(2.0f));
    object_location[1] = object_location[1] + best_guess_translation[object_idx * 3 + 1]*(0.03f/std::sqrt(2.0f));
    object_location[2] = object_location[2] + best_guess_translation[object_idx * 3 + 2]*(0.03f/std::sqrt(2.0f));

    // Normalize quaternion 
    const float best_guess_quaternion_norm = 1.0f / sqrt(best_guess_quaternion[object_idx * 4 + 0] * best_guess_quaternion[object_idx * 4 + 0] + best_guess_quaternion[object_idx * 4 + 1] * best_guess_quaternion[object_idx * 4 + 1] + best_guess_quaternion[object_idx * 4 + 2] * best_guess_quaternion[object_idx * 4 + 2] + best_guess_quaternion[object_idx * 4 + 3] * best_guess_quaternion[object_idx * 4 + 3]);
    best_guess_quaternion[object_idx * 4 + 0] *= best_guess_quaternion_norm;
    best_guess_quaternion[object_idx * 4 + 1] *= best_guess_quaternion_norm;
    best_guess_quaternion[object_idx * 4 + 2] *= best_guess_quaternion_norm;
    best_guess_quaternion[object_idx * 4 + 3] *= best_guess_quaternion_norm;
    std::cout << "Quaternion (normalized): " << best_guess_quaternion[object_idx * 4 + 0] << " " << best_guess_quaternion[object_idx * 4 + 1] << " " << best_guess_quaternion[object_idx * 4 + 2] << " " << best_guess_quaternion[object_idx * 4 + 3] << std::endl;

    // Convert quaternion to pose
    float object_pose[16];
    object_pose[0 * 4 + 0] = 1.0f - 2.0f * best_guess_quaternion[object_idx * 4 + 2] * best_guess_quaternion[object_idx * 4 + 2] - 2.0f * best_guess_quaternion[object_idx * 4 + 3] * best_guess_quaternion[object_idx * 4 + 3];
    object_pose[0 * 4 + 1] = 2.0f * best_guess_quaternion[object_idx * 4 + 1] * best_guess_quaternion[object_idx * 4 + 2] - 2.0f * best_guess_quaternion[object_idx * 4 + 3] * best_guess_quaternion[object_idx * 4 + 0];
    object_pose[0 * 4 + 2] = 2.0f * best_guess_quaternion[object_idx * 4 + 1] * best_guess_quaternion[object_idx * 4 + 3] + 2.0f * best_guess_quaternion[object_idx * 4 + 2] * best_guess_quaternion[object_idx * 4 + 0];
    object_pose[0 * 4 + 3] = object_location[0];
    object_pose[1 * 4 + 0] = 2.0f * best_guess_quaternion[object_idx * 4 + 1] * best_guess_quaternion[object_idx * 4 + 2] + 2.0f * best_guess_quaternion[object_idx * 4 + 3] * best_guess_quaternion[object_idx * 4 + 0];
    object_pose[1 * 4 + 1] = 1.0f - 2.0f * best_guess_quaternion[object_idx * 4 + 1] * best_guess_quaternion[object_idx * 4 + 1] - 2.0f * best_guess_quaternion[object_idx * 4 + 3] * best_guess_quaternion[object_idx * 4 + 3];
    object_pose[1 * 4 + 2] = 2.0f * best_guess_quaternion[object_idx * 4 + 2] * best_guess_quaternion[object_idx * 4 + 3] - 2.0f * best_guess_quaternion[object_idx * 4 + 1] * best_guess_quaternion[object_idx * 4 + 0];
    object_pose[1 * 4 + 3] = object_location[1];
    object_pose[2 * 4 + 0] = 2.0f * best_guess_quaternion[object_idx * 4 + 1] * best_guess_quaternion[object_idx * 4 + 3] - 2.0f * best_guess_quaternion[object_idx * 4 + 2] * best_guess_quaternion[object_idx * 4 + 0];
    object_pose[2 * 4 + 1] = 2.0f * best_guess_quaternion[object_idx * 4 + 2] * best_guess_quaternion[object_idx * 4 + 3] + 2.0f * best_guess_quaternion[object_idx * 4 + 1] * best_guess_quaternion[object_idx * 4 + 0];
    object_pose[2 * 4 + 2] = 1.0f - 2.0f * best_guess_quaternion[object_idx * 4 + 1] * best_guess_quaternion[object_idx * 4 + 1] - 2.0f * best_guess_quaternion[object_idx * 4 + 2] * best_guess_quaternion[object_idx * 4 + 2];
    object_pose[2 * 4 + 3] = object_location[2];
    object_pose[3 * 4 + 0] = 0.0f;
    object_pose[3 * 4 + 1] = 0.0f;
    object_pose[3 * 4 + 2] = 0.0f;
    object_pose[3 * 4 + 3] = 1.0f;

    // Display object pose
    show_object_pose(K, object_pose, curr_frame_color);

    // Show object class label
    cv::putText(curr_frame_color, "Class: " + object_names[object_idx], cv::Point(crop_x1 + 5, crop_y2 - 10), cv::FONT_HERSHEY_SIMPLEX, 0.5, cv::Scalar(0, 255, 0), 1.5);

    // // Compare against ground truth
    // if (true) {
    //   // Read ground truth object pose from file
    //   std::string gt_object_pose_filename = sequence_directory + "/object.pose.txt";
    //   std::vector<float> gt_object_pose_raw = load_matrix_from_file(gt_object_pose_filename, 4, 4);
    //   float * gt_object_pose_arr = &gt_object_pose_raw[0];

    //   // Compute ground truth object pose w.r.t. current camera pose
    //   std::string gt_cam_pose_filename = sequence_directory + "/" + frame_prefix + ".pose.txt";
    //   std::vector<float> gt_cam_pose_raw = load_matrix_from_file(gt_cam_pose_filename, 4, 4);
    //   float * gt_cam_pose_arr = &gt_cam_pose_raw[0];
    //   float gt_cam_pose_inv[16] = {0};
    //   invert_matrix(gt_cam_pose_arr, gt_cam_pose_inv);
    //   float gt_object_pose[16] = {0};
    //   multiply_matrix(gt_cam_pose_inv, gt_object_pose_arr, gt_object_pose);

    //   // Compute center of ground truth object in 3D camera coordinates
    //   float gt_object_center_cam[3] = {0};
    //   for (int i = 0; i < 3; i++)
    //     gt_object_center_cam[i] = gt_object_pose[i * 4 + 3];

    //   // Convert ground truth pose from rotation matrix to quaternion
    //   float trace = gt_object_pose[0 * 4 + 0] + gt_object_pose[1 * 4 + 1] + gt_object_pose[2 * 4 + 2]; // I removed + 1.0f; see discussion with Ethan
    //   float gt_object_pose_quaternion[4];
    //   if ( trace > 0 ) { // I changed M_EPSILON to 0
    //     float s = 0.5f / sqrtf(trace + 1.0f);
    //     gt_object_pose_quaternion[0] = 0.25f / s;
    //     gt_object_pose_quaternion[1] = ( gt_object_pose[2 * 4 + 1] - gt_object_pose[1 * 4 + 2] ) * s;
    //     gt_object_pose_quaternion[2] = ( gt_object_pose[0 * 4 + 2] - gt_object_pose[2 * 4 + 0] ) * s;
    //     gt_object_pose_quaternion[3] = ( gt_object_pose[1 * 4 + 0] - gt_object_pose[0 * 4 + 1] ) * s;
    //   } else {
    //     if ( gt_object_pose[0 * 4 + 0] > gt_object_pose[1 * 4 + 1] && gt_object_pose[0 * 4 + 0] > gt_object_pose[2 * 4 + 2] ) {
    //       float s = 2.0f * sqrtf( 1.0f + gt_object_pose[0 * 4 + 0] - gt_object_pose[1 * 4 + 1] - gt_object_pose[2 * 4 + 2]);
    //       gt_object_pose_quaternion[0] = (gt_object_pose[2 * 4 + 1] - gt_object_pose[1 * 4 + 2] ) / s;
    //       gt_object_pose_quaternion[1] = 0.25f * s;
    //       gt_object_pose_quaternion[2] = (gt_object_pose[0 * 4 + 1] + gt_object_pose[1 * 4 + 0] ) / s;
    //       gt_object_pose_quaternion[3] = (gt_object_pose[0 * 4 + 2] + gt_object_pose[2 * 4 + 0] ) / s;
    //     } else if (gt_object_pose[1 * 4 + 1] > gt_object_pose[2 * 4 + 2]) {
    //       float s = 2.0f * sqrtf( 1.0f + gt_object_pose[1 * 4 + 1] - gt_object_pose[0 * 4 + 0] - gt_object_pose[2 * 4 + 2]);
    //       gt_object_pose_quaternion[0] = (gt_object_pose[0 * 4 + 2] - gt_object_pose[2 * 4 + 0] ) / s;
    //       gt_object_pose_quaternion[1] = (gt_object_pose[0 * 4 + 1] + gt_object_pose[1 * 4 + 0] ) / s;
    //       gt_object_pose_quaternion[2] = 0.25f * s;
    //       gt_object_pose_quaternion[3] = (gt_object_pose[1 * 4 + 2] + gt_object_pose[2 * 4 + 1] ) / s;
    //     } else {
    //       float s = 2.0f * sqrtf( 1.0f + gt_object_pose[2 * 4 + 2] - gt_object_pose[0 * 4 + 0] - gt_object_pose[1 * 4 + 1] );
    //       gt_object_pose_quaternion[0] = (gt_object_pose[1 * 4 + 0] - gt_object_pose[0 * 4 + 1] ) / s;
    //       gt_object_pose_quaternion[1] = (gt_object_pose[0 * 4 + 2] + gt_object_pose[2 * 4 + 0] ) / s;
    //       gt_object_pose_quaternion[2] = (gt_object_pose[1 * 4 + 2] + gt_object_pose[2 * 4 + 1] ) / s;
    //       gt_object_pose_quaternion[3] = 0.25f * s;
    //     }
    //   }

    //   float obj_dist = sqrtf((gt_object_center_cam[0] - object_location[0]) * (gt_object_center_cam[0] - object_location[0]) +
    //                          (gt_object_center_cam[1] - object_location[1]) * (gt_object_center_cam[1] - object_location[1]) +
    //                          (gt_object_center_cam[2] - object_location[2]) * (gt_object_center_cam[2] - object_location[2]));

    //   if (obj_dist < 0.02f)
    //     cv::putText(curr_frame_color, "Detection: < 2cm conf: " + std::to_string(highest_class_scores[object_idx]), cv::Point(crop_x1 + 5, crop_y2 - 50), cv::FONT_HERSHEY_SIMPLEX, 0.5, cv::Scalar(0, 255, 0), 1.5);
    //   else
    //     cv::putText(curr_frame_color, "Detection: > 2cm conf: " + std::to_string(highest_class_scores[object_idx]), cv::Point(crop_x1 + 5, crop_y2 - 50), cv::FONT_HERSHEY_SIMPLEX, 0.5, cv::Scalar(0, 0, 255), 1.5);

    //   cv::putText(curr_frame_color, "PR: " + std::to_string(best_guess_quaternion[object_idx * 4 + 0]) + " " + std::to_string(best_guess_quaternion[object_idx * 4 + 1]) + " " + std::to_string(best_guess_quaternion[object_idx * 4 + 2]) + " " + std::to_string(best_guess_quaternion[object_idx * 4 + 3]), cv::Point(crop_x1 + 5, crop_y2 - 30), cv::FONT_HERSHEY_SIMPLEX, 0.5, cv::Scalar(0, 255, 0), 1.5);
    //   cv::putText(curr_frame_color, "GT: " + std::to_string(gt_object_pose_quaternion[0]) + " " + std::to_string(gt_object_pose_quaternion[1]) + " " + std::to_string(gt_object_pose_quaternion[2]) + " " + std::to_string(gt_object_pose_quaternion[3]), cv::Point(crop_x1 + 5, crop_y2 - 10), cv::FONT_HERSHEY_SIMPLEX, 0.5, cv::Scalar(0, 255, 0), 1.5);

    // }
  }

  cv::namedWindow("Object Poses", CV_WINDOW_AUTOSIZE);
  cv::imshow("Object Poses", curr_frame_color);
  cv::waitKey(100);

  // Save display result
  std::string results_directory = sequence_directory + "/results." + model_idx;
  sys_command("mkdir -p " + results_directory);
  cv::imwrite(results_directory + "/" + frame_prefix + ".result.png", curr_frame_color);

  // Re-alloc and re-init TSDF on GPU
  num_blocks = vox_size[2];
  num_threads = vox_size[1];
  reset_vox_whole_GPU<<<num_blocks,num_threads>>>(d_vox_size, d_vox_tsdf, d_vox_weight);
  // memset(vox_weight, 0, sizeof(float) * vox_size[0] * vox_size[1] * vox_size[2]);
  // for (int i = 0; i < vox_size[0] * vox_size[1] * vox_size[2]; i++)
  //   vox_tsdf[i] = 1.0f;
  // kCheckCUDA(__LINE__, hipMalloc(&d_vox_tsdf, vox_size[0] * vox_size[1] * vox_size[2] * sizeof(float)));
  // kCheckCUDA(__LINE__, hipMalloc(&d_vox_weight, vox_size[0] * vox_size[1] * vox_size[2] * sizeof(float)));
  // kCheckCUDA(__LINE__, hipMemcpy(d_vox_tsdf, vox_tsdf, vox_size[0] * vox_size[1] * vox_size[2] * sizeof(float), hipMemcpyHostToDevice));
  // kCheckCUDA(__LINE__, hipMemcpy(d_vox_weight, vox_weight, vox_size[0] * vox_size[1] * vox_size[2] * sizeof(float), hipMemcpyHostToDevice));

  // Clear other excess memory
  // delete [] class_score_raw;
  // delete [] axis_score_raw;
  // delete [] angle_score_raw;
  delete [] hypothesis_locations;
  delete [] hypothesis_labels;
  delete [] hypothesis_crop_2D;
}

int main(int argc, char **argv) {



  init_fusion_GPU();

  init_marvin();
  

  // tic();
  // detect("data/train/expo/000000","frame-000000");
  // toc();

  // // List RGB-D sequences
  // std::string object_directory = "data/train/expo";
  // std::vector<std::string> sequence_names;
  // get_files_in_directory(object_directory, sequence_names, "");
  // int rand_sequence_idx = (int)floor(gen_random_float(0, (float)sequence_names.size()));

  // for (int sequence_idx = 0; sequence_idx < sequence_names.size(); sequence_idx++) {
  //   std::string curr_sequence_name = sequence_names[sequence_idx];
  //   std::string curr_sequence_directory = object_directory + "/" + curr_sequence_name;

  //   // List RGB-D frames
  //   std::vector<std::string> frame_names;
  //   get_files_in_directory(curr_sequence_directory, frame_names, ".color.png");
  //   for (int frame_idx = 0; frame_idx < frame_names.size(); frame_idx++) {
  //     std::string curr_frame_name = frame_names[frame_idx];
  //     curr_frame_name = curr_frame_name.substr(0, curr_frame_name.length() - 10);
  //     tic();
  //     detect(curr_sequence_directory,curr_frame_name);
  //     toc();
  //   }
  // }


  std::string curr_sequence_directory = "data/train/expo/000004";
  // List RGB-D frames
  std::vector<std::string> frame_names;
  get_files_in_directory(curr_sequence_directory, frame_names, ".color.png");
  for (int frame_idx = 0; frame_idx < frame_names.size(); frame_idx++) {
    std::string curr_frame_name = frame_names[frame_idx];
    curr_frame_name = curr_frame_name.substr(0, curr_frame_name.length() - 10);
    tic();
    detect(curr_sequence_directory,curr_frame_name);
    toc();
  }








  return 0;
}



