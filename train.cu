#include "hip/hip_runtime.h"
#include "util/util.hpp"
#include <opencv2/opencv.hpp>

////////////////////////////////////////////////////////////////////////////////

void get_frustum_bounds(float* K, std::vector<std::vector<float>> &extrinsic_poses, int base_frame, int curr_frame, float* camera_relative_pose, float* view_bounds,
                        float vox_unit, int* vox_size, float* vox_range_cam) {

  // Use two extrinsic matrices to compute relative rotations between current frame and first frame
  std::vector<float> ex_pose1 = extrinsic_poses[base_frame];
  std::vector<float> ex_pose2 = extrinsic_poses[curr_frame];

  float * ex_mat1 = &ex_pose1[0];
  float * ex_mat2 = &ex_pose2[0];

  float ex_mat1_inv[16] = {0};
  invert_matrix(ex_mat1, ex_mat1_inv);
  multiply_matrix(ex_mat1_inv, ex_mat2, camera_relative_pose);

  // Init cam view frustum
  float max_depth = 0.8;
  float cam_view_frustum[15] =
  { 0, -320 * max_depth / K[0], -320 * max_depth / K[0], 320 * max_depth / K[0],  320 * max_depth / K[0],
    0, -240 * max_depth / K[0],  240 * max_depth / K[0], 240 * max_depth / K[0], -240 * max_depth / K[0],
    0,               max_depth,               max_depth,              max_depth,              max_depth
  };

  // Rotate cam view frustum wrt Rt
  for (int i = 0; i < 5; i++) {
    float tmp_arr[3] = {0};
    tmp_arr[0] = camera_relative_pose[0 * 4 + 0] * cam_view_frustum[0 + i] + camera_relative_pose[0 * 4 + 1] * cam_view_frustum[5 + i] + camera_relative_pose[0 * 4 + 2] * cam_view_frustum[2 * 5 + i];
    tmp_arr[1] = camera_relative_pose[1 * 4 + 0] * cam_view_frustum[0 + i] + camera_relative_pose[1 * 4 + 1] * cam_view_frustum[5 + i] + camera_relative_pose[1 * 4 + 2] * cam_view_frustum[2 * 5 + i];
    tmp_arr[2] = camera_relative_pose[2 * 4 + 0] * cam_view_frustum[0 + i] + camera_relative_pose[2 * 4 + 1] * cam_view_frustum[5 + i] + camera_relative_pose[2 * 4 + 2] * cam_view_frustum[2 * 5 + i];
    cam_view_frustum[0 * 5 + i] = tmp_arr[0] + camera_relative_pose[3];
    cam_view_frustum[1 * 5 + i] = tmp_arr[1] + camera_relative_pose[7];
    cam_view_frustum[2 * 5 + i] = tmp_arr[2] + camera_relative_pose[11];
  }

  // Compute frustum endpoints
  float range2test[3][2] = {0};
  for (int i = 0; i < 3; i++) {
    range2test[i][0] = *std::min_element(&cam_view_frustum[i * 5], &cam_view_frustum[i * 5] + 5);
    range2test[i][1] = *std::max_element(&cam_view_frustum[i * 5], &cam_view_frustum[i * 5] + 5);
  }

  // Compute frustum bounds wrt volume
  for (int i = 0; i < 3; i++) {
    view_bounds[i * 2 + 0] = std::max(0.0f, std::floor((range2test[i][0] - vox_range_cam[i * 2 + 0]) / vox_unit));
    view_bounds[i * 2 + 1] = std::min((float)(vox_size[i]), std::ceil((range2test[i][1] - vox_range_cam[i * 2 + 0]) / vox_unit + 1));
  }
}

////////////////////////////////////////////////////////////////////////////////

void save_volume_to_ply(const std::string &file_name, int* vox_size, float* vox_tsdf, float* vox_weight) {
  float tsdf_threshold = 0.2f;
  float weight_threshold = 1.0f;
  // float radius = 5.0f;

  // Count total number of points in point cloud
  int num_points = 0;
  for (int i = 0; i < vox_size[0] * vox_size[1] * vox_size[2]; i++)
    if (std::abs(vox_tsdf[i]) < tsdf_threshold && vox_weight[i] >= weight_threshold)
      num_points++;

  // Create header for ply file
  FILE *fp = fopen(file_name.c_str(), "w");
  fprintf(fp, "ply\n");
  fprintf(fp, "format binary_little_endian 1.0\n");
  fprintf(fp, "element vertex %d\n", num_points);
  fprintf(fp, "property float x\n");
  fprintf(fp, "property float y\n");
  fprintf(fp, "property float z\n");
  fprintf(fp, "end_header\n");

  // Create point cloud content for ply file
  for (int i = 0; i < vox_size[0] * vox_size[1] * vox_size[2]; i++) {

    // If TSDF value of voxel is less than some threshold, add voxel coordinates to point cloud
    if (std::abs(vox_tsdf[i]) < tsdf_threshold && vox_weight[i] >= weight_threshold) {

      // Compute voxel indices in int for higher positive number range
      int z = floor(i / (vox_size[0] * vox_size[1]));
      int y = floor((i - (z * vox_size[0] * vox_size[1])) / vox_size[0]);
      int x = i - (z * vox_size[0] * vox_size[1]) - (y * vox_size[0]);

      // Convert voxel indices to float, and save coordinates to ply file
      float float_x = (float) x;
      float float_y = (float) y;
      float float_z = (float) z;
      fwrite(&float_x, sizeof(float), 1, fp);
      fwrite(&float_y, sizeof(float), 1, fp);
      fwrite(&float_z, sizeof(float), 1, fp);
    }
  }
  fclose(fp);
}

////////////////////////////////////////////////////////////////////////////////

__global__
void integrate(float* tmp_K, unsigned short* tmp_depth_data, float* tmp_view_bounds, float* tmp_camera_relative_pose,
               float tmp_vox_unit, float tmp_vox_mu, int* tmp_vox_size, float* tmp_vox_range_cam, float* tmp_vox_tsdf, float* tmp_vox_weight) {

  int z = blockIdx.x;
  int y = threadIdx.x;

  if (z < (int)tmp_view_bounds[2 * 2 + 0] || z >= (int)tmp_view_bounds[2 * 2 + 1])
    return;
  if (y < (int)tmp_view_bounds[1 * 2 + 0] || y >= (int)tmp_view_bounds[1 * 2 + 1])
    return;
  for (int x = tmp_view_bounds[0 * 2 + 0]; x < tmp_view_bounds[0 * 2 + 1]; x++) {

    // grid to world coords
    float tmp_pos[3] = {0};
    tmp_pos[0] = (x + 1) * tmp_vox_unit + tmp_vox_range_cam[0 * 2 + 0];
    tmp_pos[1] = (y + 1) * tmp_vox_unit + tmp_vox_range_cam[1 * 2 + 0];
    tmp_pos[2] = (z + 1) * tmp_vox_unit + tmp_vox_range_cam[2 * 2 + 0];

    // transform
    float tmp_arr[3] = {0};
    tmp_arr[0] = tmp_pos[0] - tmp_camera_relative_pose[3];
    tmp_arr[1] = tmp_pos[1] - tmp_camera_relative_pose[7];
    tmp_arr[2] = tmp_pos[2] - tmp_camera_relative_pose[11];
    tmp_pos[0] = tmp_camera_relative_pose[0 * 4 + 0] * tmp_arr[0] + tmp_camera_relative_pose[1 * 4 + 0] * tmp_arr[1] + tmp_camera_relative_pose[2 * 4 + 0] * tmp_arr[2];
    tmp_pos[1] = tmp_camera_relative_pose[0 * 4 + 1] * tmp_arr[0] + tmp_camera_relative_pose[1 * 4 + 1] * tmp_arr[1] + tmp_camera_relative_pose[2 * 4 + 1] * tmp_arr[2];
    tmp_pos[2] = tmp_camera_relative_pose[0 * 4 + 2] * tmp_arr[0] + tmp_camera_relative_pose[1 * 4 + 2] * tmp_arr[1] + tmp_camera_relative_pose[2 * 4 + 2] * tmp_arr[2];
    if (tmp_pos[2] <= 0)
      continue;

    int px = roundf(tmp_K[0] * (tmp_pos[0] / tmp_pos[2]) + tmp_K[2]);
    int py = roundf(tmp_K[4] * (tmp_pos[1] / tmp_pos[2]) + tmp_K[5]);
    if (px < 1 || px > 640 || py < 1 || py > 480)
      continue;

    float p_depth = *(tmp_depth_data + (py - 1) * 640 + (px - 1)) / 1000.f;
    if (p_depth < 0.2 || p_depth > 0.8)
      continue;
    if (roundf(p_depth * 1000.0f) == 0)
      continue;

    float eta = (p_depth - tmp_pos[2]) * sqrtf(1 + powf((tmp_pos[0] / tmp_pos[2]), 2) + powf((tmp_pos[1] / tmp_pos[2]), 2));
    if (eta <= -tmp_vox_mu)
      continue;

    // Integrate
    int volumeIDX = z * tmp_vox_size[0] * tmp_vox_size[1] + y * tmp_vox_size[0] + x;
    float sdf = fmin(1.0f, eta / tmp_vox_mu);
    float w_old = tmp_vox_weight[volumeIDX];
    float w_new = w_old + 1.0f;
    tmp_vox_weight[volumeIDX] = w_new;
    tmp_vox_tsdf[volumeIDX] = (tmp_vox_tsdf[volumeIDX] * w_old + sdf) / w_new;
  }
}

////////////////////////////////////////////////////////////////////////////////

void vol2bin() {
  // Write data to binary file
  // std::string volume_filename = "volume.tsdf.bin";
  // std::ofstream out_file(volume_filename, std::ios::binary | std::ios::out);
  // for (int i = 0; i < vox_size[0] * vox_size[1] * vox_size[2]; i++)
  //   out_file.write((char*)&vox_tsdf[i], sizeof(float));
  // out_file.close();
}

////////////////////////////////////////////////////////////////////////////////

void FatalError(const int lineNumber = 0) {
  std::cerr << "FatalError";
  if (lineNumber != 0) std::cerr << " at LINE " << lineNumber;
  std::cerr << ". Program Terminated." << std::endl;
  hipDeviceReset();
  exit(EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////

void checkCUDA(const int lineNumber, hipError_t status) {
  if (status != hipSuccess) {
    std::cerr << "CUDA failure at LINE " << lineNumber << ": " << status << std::endl;
    FatalError();
  }
}

////////////////////////////////////////////////////////////////////////////////

void show_object_pose(float* K, float* object_pose, cv::Mat& display_frame) {

  // Compute center of ground truth object in 3D camera coordinates
  float object_center_display_3D[3] = {0};
  for (int i = 0; i < 3; i++)
    object_center_display_3D[i] = object_pose[i * 4 + 3];

  // Compute axis endpoints of ground truth object pose in 3D camera coordinates
  float object_pose_display_3D[18] = {0};
  for (int i = 0; i < 3; i++) {
    object_pose_display_3D[0 * 6 + i * 2 + 0] = object_pose[0 * 4 + 3] - object_pose[0 * 4 + i] * 0.1f;
    object_pose_display_3D[0 * 6 + i * 2 + 1] = object_pose[0 * 4 + 3] + object_pose[0 * 4 + i] * 0.1f;
    object_pose_display_3D[1 * 6 + i * 2 + 0] = object_pose[1 * 4 + 3] - object_pose[1 * 4 + i] * 0.1f;
    object_pose_display_3D[1 * 6 + i * 2 + 1] = object_pose[1 * 4 + 3] + object_pose[1 * 4 + i] * 0.1f;
    object_pose_display_3D[2 * 6 + i * 2 + 0] = object_pose[2 * 4 + 3] - object_pose[2 * 4 + i] * 0.1f;
    object_pose_display_3D[2 * 6 + i * 2 + 1] = object_pose[2 * 4 + 3] + object_pose[2 * 4 + i] * 0.1f;
  }

  // Project endpoints of ground truth object pose axis from 3D to 2D
  float object_pose_display_2D[12] = {0};
  for (int i = 0; i < 6; i++) {
    object_pose_display_2D[0 * 6 + i] = (object_pose_display_3D[0 * 6 + i] * K[0]) / (object_pose_display_3D[2 * 6 + i]) + K[2];
    object_pose_display_2D[1 * 6 + i] = (object_pose_display_3D[1 * 6 + i] * K[4]) / (object_pose_display_3D[2 * 6 + i]) + K[5];
  }
  // for (int i = 0; i < 12; i++)
  //   std::cout << object_pose_display_2D[i] << std::endl;

  // Project center of ground truth object from 3D to 2D
  float object_center_display_2D[2] = {0};
  object_center_display_2D[0] = (object_center_display_3D[0] * K[0]) / (object_center_display_3D[2]) + K[2];
  object_center_display_2D[1] = (object_center_display_3D[1] * K[4]) / (object_center_display_3D[2]) + K[5];
  // for (int i = 0; i < 12; i++)
  //   std::cout << object_pose_display_2D[i] << std::endl;

  // Display ground truth object pose
  cv::line(display_frame, cv::Point(object_pose_display_2D[0], object_pose_display_2D[6]), cv::Point(object_pose_display_2D[1], object_pose_display_2D[7]), cv::Scalar(0, 0, 255), 3);
  cv::line(display_frame, cv::Point(object_pose_display_2D[2], object_pose_display_2D[8]), cv::Point(object_pose_display_2D[3], object_pose_display_2D[9]), cv::Scalar(0, 255, 0), 3);
  cv::line(display_frame, cv::Point(object_pose_display_2D[4], object_pose_display_2D[10]), cv::Point(object_pose_display_2D[5], object_pose_display_2D[11]), cv::Scalar(255, 0, 0), 3);
  cv::circle(display_frame, cv::Point(object_center_display_2D[0], object_center_display_2D[1]), 6, cv::Scalar(0, 255, 255), -1);
  cv::namedWindow("Object Pose", CV_WINDOW_AUTOSIZE);
  cv::imshow("Object Pose", display_frame);
}

////////////////////////////////////////////////////////////////////////////////
// Fusion: always keep a TSDF volume active in GPU

// TSDF volume in CPU memory
float vox_unit;
float vox_mu_grid;
float vox_mu;
int vox_size[3];
float vox_range_cam[6];
float * vox_tsdf;
float * vox_weight;

// TSDF volume in GPU memory
int * d_vox_size;
float * d_vox_tsdf;
float * d_vox_weight;

// Fusion params in GPU memory
float * d_K;
unsigned short * d_depth_data;
float * d_view_bounds;
float * d_camera_relative_pose;
float * d_vox_range_cam;

// Initialize existing TSDF volume in GPU memory
__global__
void reset_vox_GPU(int* tmp_vox_size, float* tmp_vox_tsdf, float* tmp_vox_weight) {
  int z = blockIdx.x;
  int y = threadIdx.x;
  for (int x = 0; x < tmp_vox_size[0]; x++) {
    tmp_vox_tsdf[z * tmp_vox_size[0] * tmp_vox_size[1] + y * tmp_vox_size[0] + x] = 1.0f;
    tmp_vox_weight[z * tmp_vox_size[0] * tmp_vox_size[1] + y * tmp_vox_size[0] + x] = 0;
  }
}

////////////////////////////////////////////////////////////////////////////////

// Initialize TSDF volume and fusion params
void init_fusion_GPU() {

  // Init voxel volume params
  vox_unit = 0.005;
  vox_mu_grid = 5;
  vox_mu = vox_unit * vox_mu_grid;
  vox_size[0] = 512;
  vox_size[1] = 512;
  vox_size[2] = 512;
  vox_range_cam[0 * 2 + 0] = -(float)(vox_size[0]) * vox_unit / 2;
  vox_range_cam[0 * 2 + 1] = vox_range_cam[0 * 2 + 0] + (float)(vox_size[0]) * vox_unit;
  vox_range_cam[1 * 2 + 0] = -(float)(vox_size[1]) * vox_unit / 2;
  vox_range_cam[1 * 2 + 1] = vox_range_cam[1 * 2 + 0] + (float)(vox_size[1]) * vox_unit;
  vox_range_cam[2 * 2 + 0] = -50.0f * vox_unit;
  vox_range_cam[2 * 2 + 1] = vox_range_cam[2 * 2 + 0] + (float)(vox_size[2]) * vox_unit;
  vox_tsdf = new float[vox_size[0] * vox_size[1] * vox_size[2]];
  vox_weight = new float[vox_size[0] * vox_size[1] * vox_size[2]];
  memset(vox_weight, 0, sizeof(float) * vox_size[0] * vox_size[1] * vox_size[2]);
  memset(vox_tsdf, 0, sizeof(float) * vox_size[0] * vox_size[1] * vox_size[2]);
  // for (int i = 0; i < vox_size[0] * vox_size[1] * vox_size[2]; i++)
  //   vox_tsdf[i] = 1.0f;

  // Copy voxel volume to GPU
  hipMalloc(&d_vox_size, 3 * sizeof(float));
  hipMalloc(&d_vox_tsdf, vox_size[0] * vox_size[1] * vox_size[2] * sizeof(float));
  hipMalloc(&d_vox_weight, vox_size[0] * vox_size[1] * vox_size[2] * sizeof(float));
  checkCUDA(__LINE__, hipGetLastError());
  hipMemcpy(d_vox_size, vox_size, 3 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vox_tsdf, vox_tsdf, vox_size[0] * vox_size[1] * vox_size[2] * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vox_weight, vox_weight, vox_size[0] * vox_size[1] * vox_size[2] * sizeof(float), hipMemcpyHostToDevice);
  checkCUDA(__LINE__, hipGetLastError());

  // Init volume in GPU
  int CUDA_NUM_BLOCKS = vox_size[2];
  int CUDA_NUM_THREADS = vox_size[1];
  reset_vox_GPU <<< CUDA_NUM_BLOCKS, CUDA_NUM_THREADS >>>(d_vox_size, d_vox_tsdf, d_vox_weight);
  checkCUDA(__LINE__, hipGetLastError());

  // Allocate GPU to hold fusion params
  hipMalloc(&d_K, 9 * sizeof(float));
  hipMalloc(&d_depth_data, 480 * 640 * sizeof(unsigned short));
  hipMalloc(&d_view_bounds, 6 * sizeof(float));
  hipMalloc(&d_camera_relative_pose, 16 * sizeof(float));
  hipMalloc(&d_vox_range_cam, 6 * sizeof(float));
  checkCUDA(__LINE__, hipGetLastError());
}

////////////////////////////////////////////////////////////////////////////////

__global__
void gen_hypothesis_labels(int num_hypothesis, unsigned short* tmp_hypothesis_locations, char* tmp_hypothesis_labels, unsigned short* tmp_hypothesis_crop_2D, float* tmp_object_center_cam, float* tmp_K, float tmp_vox_unit, int* tmp_vox_size, float* tmp_vox_range_cam, float* tmp_vox_tsdf) {

  // Check kernel index
  int hypothesis_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (hypothesis_idx >= num_hypothesis)
    return;

  float tsdf_surface_threshold = 0.2f;
  int cube_dim = 30;

  // Fetch hypothesis location
  int x = (int)tmp_hypothesis_locations[0 * num_hypothesis + hypothesis_idx];
  int y = (int)tmp_hypothesis_locations[1 * num_hypothesis + hypothesis_idx];
  int z = (int)tmp_hypothesis_locations[2 * num_hypothesis + hypothesis_idx];

  // Check cube occupancy
  int cube_occ = 0;
  for (int i = -15; i < 15; i++)
    for (int j = -15; j < 15; j++)
      for (int k = -15; k < 15; k++) {
        int vox_idx = (z + k) * tmp_vox_size[0] * tmp_vox_size[1] + (y + j) * tmp_vox_size[0] + (x + i);
        if (tmp_vox_tsdf[vox_idx] < tsdf_surface_threshold)
          cube_occ++;
      }

  // Skip near empty cubes
  if (cube_occ < cube_dim * cube_dim / 2)
    return;

  // Convert cube location from grid to camera coordinates
  float x_cam = ((float)x + 1) * tmp_vox_unit + tmp_vox_range_cam[0 * 2 + 0];
  float y_cam = ((float)y + 1) * tmp_vox_unit + tmp_vox_range_cam[1 * 2 + 0];
  float z_cam = ((float)z + 1) * tmp_vox_unit + tmp_vox_range_cam[2 * 2 + 0];

  // If cube 2D projection is not in image bounds, cube is invalid
  float cube_rad = ((float) cube_dim) * tmp_vox_unit / 2;
  float cube_front[12] = {(x_cam + cube_rad), (x_cam + cube_rad), (x_cam - cube_rad), (x_cam - cube_rad),
                          (y_cam + cube_rad), (y_cam - cube_rad), (y_cam - cube_rad), (y_cam + cube_rad),
                          (z_cam - cube_rad), (z_cam - cube_rad), (z_cam - cube_rad), (z_cam - cube_rad)
                         };
  float cube_front_2D[8] = {};
  for (int i = 0; i < 4; i++) {
    cube_front_2D[0 * 4 + i] = cube_front[0 * 4 + i] * tmp_K[0] / cube_front[2 * 4 + i] + tmp_K[2];
    cube_front_2D[1 * 4 + i] = cube_front[1 * 4 + i] * tmp_K[4] / cube_front[2 * 4 + i] + tmp_K[5];
  }
  for (int i = 0; i < 8; i++)
    cube_front_2D[i] = roundf(cube_front_2D[i]);
  if (fmin(fmin(cube_front_2D[0], cube_front_2D[1]), fmin(cube_front_2D[2], cube_front_2D[3])) < 0 ||
      fmax(fmax(cube_front_2D[0], cube_front_2D[1]), fmax(cube_front_2D[2], cube_front_2D[3])) >= 640 ||
      fmin(fmin(cube_front_2D[4], cube_front_2D[5]), fmin(cube_front_2D[6], cube_front_2D[7])) < 0 ||
      fmax(fmax(cube_front_2D[4], cube_front_2D[5]), fmax(cube_front_2D[6], cube_front_2D[7])) >= 480)
    return;

  // Check distance between center of cube to center of object bbox
  float obj_dist = sqrtf((tmp_object_center_cam[0] - x_cam) * (tmp_object_center_cam[0] - x_cam) +
                         (tmp_object_center_cam[1] - y_cam) * (tmp_object_center_cam[1] - y_cam) +
                         (tmp_object_center_cam[2] - z_cam) * (tmp_object_center_cam[2] - z_cam));

  // Save label (positive case if dist to ground truth object center < some threshold)
  if (obj_dist < 0.01f)
    tmp_hypothesis_labels[hypothesis_idx] = (char)1;
  else
    tmp_hypothesis_labels[hypothesis_idx] = (char)2;

  // Save 2D patch of cube's 2D project to image
  tmp_hypothesis_crop_2D[0 * num_hypothesis + hypothesis_idx] = (unsigned short)roundf(cube_front_2D[2]);
  tmp_hypothesis_crop_2D[1 * num_hypothesis + hypothesis_idx] = (unsigned short)roundf(cube_front_2D[6]);
  tmp_hypothesis_crop_2D[2 * num_hypothesis + hypothesis_idx] = (unsigned short)roundf(cube_front_2D[1] - cube_front_2D[2]);
  tmp_hypothesis_crop_2D[3 * num_hypothesis + hypothesis_idx] = (unsigned short)roundf(cube_front_2D[4] - cube_front_2D[6]);
}

////////////////////////////////////////////////////////////////////////////////

__global__
void check_valid_hypothesis_loc(char* tmp_is_grid_loc_valid, int* tmp_vox_size, float* tmp_vox_tsdf) {

  int z = blockIdx.x;
  int y = threadIdx.x;
  if (z < 15 || z >= (tmp_vox_size[2] - 15))
    return;
  if (y < 15 || y >= (tmp_vox_size[1] - 15))
    return;

  float tsdf_threshold = 0.2f;
  int cube_dim = 30;

  for (int x = 15; x < (tmp_vox_size[0] - 15); x++) {
    int loc_idx = blockIdx.x * tmp_vox_size[1] * tmp_vox_size[0] + threadIdx.x * tmp_vox_size[0] + x;

    int cube_occ = 0;
    for (int i = -15; i < 15; i++)
      for (int j = -15; j < 15; j++)
        for (int k = -15; k < 15; k++) {
          int vox_idx = (z + k) * tmp_vox_size[0] * tmp_vox_size[1] + (y + j) * tmp_vox_size[0] + (x + i);
          if (tmp_vox_tsdf[vox_idx] < tsdf_threshold)
            cube_occ++;
        }

    // Non-empty cubes are valid cubes
    if (cube_occ > cube_dim * cube_dim / 2)
      tmp_is_grid_loc_valid[loc_idx] = 1;
  }
}

////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv) {

  init_fusion_GPU();

  std::string data_directory = "data/train";
  std::string object_name = "glue";
  std::string object_directory = data_directory + "/" + object_name;

  // Pick a random RGB-D sequence
  std::vector<std::string> sequence_names;
  get_files_in_directory(object_directory, sequence_names, "");
  int rand_sequence_idx = (int)floor(gen_random_float(0, (float)sequence_names.size()));
  std::string curr_sequence_name = sequence_names[rand_sequence_idx];
  std::string curr_sequence_directory = object_directory + "/" + curr_sequence_name;
  // std::cout << curr_sequence_directory << std::endl;

  // Pick a random RGB-D frame
  std::vector<std::string> frame_names;
  get_files_in_directory(curr_sequence_directory, frame_names, ".color.png");
  // for (int i = 0; i < frame_names.size(); i++)
  //   std::cout << frame_names[i] << std::endl;
  int rand_frame_idx = (int)floor(gen_random_float(0, (float)frame_names.size()));
  std::string curr_frame_name = frame_names[rand_frame_idx];
  curr_frame_name = curr_frame_name.substr(0, curr_frame_name.length() - 10);
  std::cout << "Preparing Training Frame: " << curr_sequence_directory << "/" << curr_frame_name << std::endl;

  // Load intrinsics (3x3 matrix)
  std::string intrinsic_filename = curr_sequence_directory + "/intrinsics.K.txt";
  std::vector<float> K_vec = load_matrix_from_file(intrinsic_filename, 3, 3);
  float K[9];
  for (int i = 0; i < 9; i++)
    K[i] = K_vec[i];
  // for (int i = 0; i < 9; i++)
  //   std::cout << K[i] << std::endl;

  // Load RGB-D frame
  std::string curr_frame_color_filename = curr_sequence_directory + "/" + curr_frame_name + ".color.png";
  cv::Mat curr_frame_color = cv::imread(curr_frame_color_filename.c_str(), 1);
  std::string curr_frame_depth_filename = curr_sequence_directory + "/" + curr_frame_name + ".depth.png";
  cv::Mat curr_frame_depth = cv::imread(curr_frame_depth_filename.c_str(), CV_LOAD_IMAGE_UNCHANGED);

  // Read ground truth object pose from file
  std::string object_pose_filename = curr_sequence_directory + "/object.pose.txt";
  std::vector<float> object_pose_raw = load_matrix_from_file(object_pose_filename, 4, 4);
  float * object_pose_arr = &object_pose_raw[0];
  // for (int i = 0; i < 12; i++)
  //   std::cout << object_pose[i] << std::endl;

  // Compute ground truth object pose w.r.t. current camera pose
  std::string curr_cam_pose_filename = curr_sequence_directory + "/" + curr_frame_name + ".pose.txt";
  std::vector<float> curr_cam_pose_raw = load_matrix_from_file(curr_cam_pose_filename, 4, 4);
  float * curr_cam_pose_arr = &curr_cam_pose_raw[0];
  // for (int i = 0; i < 16; i++)
  //   std::cout << curr_cam_pose_arr[i] << std::endl;
  float curr_cam_pose_inv[16] = {0};
  invert_matrix(curr_cam_pose_arr, curr_cam_pose_inv);
  // for (int i = 0; i < 16; i++)
  //   std::cout << curr_cam_pose_inv[i] << std::endl;
  float object_pose[16] = {0};
  multiply_matrix(curr_cam_pose_inv, object_pose_arr, object_pose);
  // for (int i = 0; i < 4; i++) {
  //   for (int j = 0; j < 4; j++)
  //     std::cout << object_pose[i * 4 + j] << " ";
  //   std::cout << std::endl;
  // }

  // Display ground truth object pose
  show_object_pose(K, object_pose, curr_frame_color);
  cv::waitKey(0);

  // Compute center of ground truth object in 3D camera coordinates
  float object_center_cam[3] = {0};
  for (int i = 0; i < 3; i++)
    object_center_cam[i] = object_pose[i * 4 + 3];

  // Convert pose from rotation matrix to axis/angle (radians) representation (x, y, z, theta)
  float object_pose_axis[3] = {0};
  float object_pose_angle = std::acos(0.5f * (object_pose[0] + object_pose[5] + object_pose[10] - 1));
  object_pose_axis[0] = (object_pose[9] - object_pose[6]) / (2 * std::sin(object_pose_angle));
  object_pose_axis[1] = (object_pose[2] - object_pose[8]) / (2 * std::sin(object_pose_angle));
  object_pose_axis[2] = (object_pose[4] - object_pose[1]) / (2 * std::sin(object_pose_angle));
  // for (int i = 0; i < 3; i++)
  //   std::cout << object_pose_axis[i] << std::endl;

  // Convert axis/angle to pose
  float object_pose_rotation[9] = {0};
  object_pose_rotation[0 * 3 + 0] = (1 - std::cos(object_pose_angle)) * object_pose_axis[0] * object_pose_axis[0] + std::cos(object_pose_angle);
  object_pose_rotation[0 * 3 + 1] = (1 - std::cos(object_pose_angle)) * object_pose_axis[0] * object_pose_axis[1] - object_pose_axis[2] * std::sin(object_pose_angle);
  object_pose_rotation[0 * 3 + 2] = (1 - std::cos(object_pose_angle)) * object_pose_axis[0] * object_pose_axis[2] + object_pose_axis[1] * std::sin(object_pose_angle);
  object_pose_rotation[1 * 3 + 0] = (1 - std::cos(object_pose_angle)) * object_pose_axis[1] * object_pose_axis[0] + object_pose_axis[2] * std::sin(object_pose_angle);
  object_pose_rotation[1 * 3 + 1] = (1 - std::cos(object_pose_angle)) * object_pose_axis[1] * object_pose_axis[1] + std::cos(object_pose_angle);
  object_pose_rotation[1 * 3 + 2] = (1 - std::cos(object_pose_angle)) * object_pose_axis[1] * object_pose_axis[2] - object_pose_axis[0] * std::sin(object_pose_angle);
  object_pose_rotation[2 * 3 + 0] = (1 - std::cos(object_pose_angle)) * object_pose_axis[2] * object_pose_axis[0] - object_pose_axis[1] *  std::sin(object_pose_angle);
  object_pose_rotation[2 * 3 + 1] = (1 - std::cos(object_pose_angle)) * object_pose_axis[2] * object_pose_axis[1] + object_pose_axis[0] * std::sin(object_pose_angle);
  object_pose_rotation[2 * 3 + 2] = (1 - std::cos(object_pose_angle)) * object_pose_axis[2] * object_pose_axis[2] + std::cos(object_pose_angle);
  // for (int i = 0; i < 3; i++) {
  //   for (int j = 0; j < 3; j++)
  //     std::cout << object_pose_rotation[i * 3 + j] << " ";
  //   std::cout << std::endl;
  // }

  // Bin axis into one of 42 bins
  float axis_sphere_bin[42 * 3] = { -0.85065, -1, -0.85065, -0.80902, -0.80902, -0.80902, -0.80902, -0.52573, -0.52573, -0.5, -0.5, -0.5, -0.5, -0.30902, -0.30902, -0.30902, -0.30902, 0, 0, 0, 0, 0, 0, 0, 0, 0.30902, 0.30902, 0.30902, 0.30902, 0.5, 0.5, 0.5, 0.5, 0.52573, 0.52573, 0.80902, 0.80902, 0.80902, 0.80902, 0.85065, 1, 0.85065,
                                    0, 0, 0, -0.5, -0.5, 0.5, 0.5, -0.85065, 0.85065, -0.30902, -0.30902, 0.30902, 0.30902, -0.80902, -0.80902, 0.80902, 0.80902, -1, -0.52573, -0.52573, 0, 0, 0.52573, 0.52573, 1, -0.80902, -0.80902, 0.80902, 0.80902, -0.30902, -0.30902, 0.30902, 0.30902, -0.85065, 0.85065, -0.5, -0.5, 0.5, 0.5, 0, 0, 0,
                                    -0.52573, 0, 0.52573, -0.30902, 0.30902, -0.30902, 0.30902, 0, 0, -0.80902, 0.80902, -0.80902, 0.80902, -0.5, 0.5, -0.5, 0.5, 0, -0.85065, 0.85065, -1, 1, -0.85065, 0.85065, 0, -0.5, 0.5, -0.5, 0.5, -0.80902, 0.80902, -0.80902, 0.80902, 0, 0, -0.30902, 0.30902, -0.30902, 0.30902, -0.52573, 0, 0.52573
                                  };
  int closest_axis_bin = 0;
  float closest_axis_dist = 100;
  for (int i = 0; i < 42; i++) {
    float curr_axis_dist = std::sqrt((axis_sphere_bin[0 * 42 + i] - object_pose_axis[0]) * (axis_sphere_bin[0 * 42 + i] - object_pose_axis[0]) +
                                     (axis_sphere_bin[1 * 42 + i] - object_pose_axis[1]) * (axis_sphere_bin[1 * 42 + i] - object_pose_axis[1]) +
                                     (axis_sphere_bin[2 * 42 + i] - object_pose_axis[2]) * (axis_sphere_bin[2 * 42 + i] - object_pose_axis[2]));
    if (curr_axis_dist < closest_axis_dist) {
      closest_axis_dist = curr_axis_dist;
      closest_axis_bin = i;
    }
  }
  // std::cout << closest_axis_bin << std::endl;
  // std::cout << closest_axis_dist << std::endl;

  // Bin angle into one of 18 bins (10 degrees)
  float closest_angle_bin = floor(object_pose_angle / (3.14159265 / 18));
  if (closest_angle_bin > 17 || closest_axis_bin > 41) {
    std::cout << "AXIS/ANGLE BINS INCORRECTLY SET UP" << std::endl;
    exit(1);
  }

  // Load image/depth/extrinsic data for current frame
  unsigned short * depth_data = (unsigned short *) malloc(480 * 640 * sizeof(unsigned short));
  for (int i = 0; i < 480 * 640; i++)
    depth_data[i] = (((unsigned short) curr_frame_depth.data[i * 2 + 1]) << 8) + ((unsigned short) curr_frame_depth.data[i * 2 + 0]);

  // Compute relative camera pose transform between current frame and base frame
  // Compute camera view frustum bounds within the voxel volume
  float camera_relative_pose[16] = {0};
  float view_bounds[6] = {0};
  std::vector<float> curr_extrinsic;
  for (int i = 0; i < 3; i++) {
    curr_extrinsic.push_back(1.0f);
    for (int i = 0; i < 4; i++) {
      curr_extrinsic.push_back(0.0f);
    }
  }
  curr_extrinsic.push_back(1.0f);
  std::vector<std::vector<float>> extrinsics;
  extrinsics.push_back(curr_extrinsic);
  get_frustum_bounds(K, extrinsics, 0, 0, camera_relative_pose, view_bounds,
                     vox_unit, vox_size, vox_range_cam);

  // Copy fusion params to GPU
  hipMemcpy(d_K, K, 9 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_depth_data, depth_data, 480 * 640 * sizeof(unsigned short), hipMemcpyHostToDevice);
  hipMemcpy(d_view_bounds, view_bounds, 6 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_camera_relative_pose, camera_relative_pose, 16 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vox_range_cam, vox_range_cam, 6 * sizeof(float), hipMemcpyHostToDevice);
  checkCUDA(__LINE__, hipGetLastError());

  // Integrate
  int CUDA_NUM_BLOCKS = vox_size[2];
  int CUDA_NUM_THREADS = vox_size[1];
  integrate <<< CUDA_NUM_BLOCKS, CUDA_NUM_THREADS >>>(d_K, d_depth_data, d_view_bounds, d_camera_relative_pose,
      vox_unit, vox_mu, d_vox_size, d_vox_range_cam, d_vox_tsdf, d_vox_weight);
  checkCUDA(__LINE__, hipGetLastError());

  // // Clear memory hold depth frame
  // free(depth_data);

  // // Reset volume in GPU
  // reset_vox_GPU <<< CUDA_NUM_BLOCKS, CUDA_NUM_THREADS >>>(d_vox_size, d_vox_tsdf, d_vox_weight);
  // checkCUDA(__LINE__, hipGetLastError());

  // Copy data back to memory
  hipMemcpy(vox_tsdf, d_vox_tsdf, vox_size[0] * vox_size[1] * vox_size[2] * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(vox_weight, d_vox_weight, vox_size[0] * vox_size[1] * vox_size[2] * sizeof(float), hipMemcpyDeviceToHost);
  checkCUDA(__LINE__, hipGetLastError());

  // for (int i = 0; i < vox_size[0] * vox_size[1] * vox_size[2]; i++)
  //   std::cout << vox_tsdf[i] << std::endl;

  // Save curr volume to file
  // std::string scene_ply_name = "volume.pointcloud.ply";
  // save_volume_to_ply(scene_ply_name, vox_size, vox_tsdf, vox_weight);

  // Compute bounding box of surface in TSDF volume
  float tsdf_surface_threshold = 0.2f;
  float grid_bounds[6] = {0};
  grid_bounds[0] = vox_size[0]; grid_bounds[2] = vox_size[1]; grid_bounds[4] = vox_size[2];
  for (int i = 0; i < vox_size[0] * vox_size[1] * vox_size[2]; i++) {
    if (std::abs(vox_tsdf[i]) < tsdf_surface_threshold) {
      float z = (float) (floor(i / (vox_size[0] * vox_size[1])));
      float y = (float) (floor((i - (z * vox_size[0] * vox_size[1])) / vox_size[0]));
      float x = (float) (i - (z * vox_size[0] * vox_size[1]) - (y * vox_size[0]));
      grid_bounds[0] = std::min(x, grid_bounds[0]); grid_bounds[1] = std::max(x, grid_bounds[1]);
      grid_bounds[2] = std::min(y, grid_bounds[2]); grid_bounds[3] = std::max(y, grid_bounds[3]);
      grid_bounds[4] = std::min(z, grid_bounds[4]); grid_bounds[5] = std::max(z, grid_bounds[5]);
    }
  }

  // Double check bounding box is not near edge of TSDF volume
  grid_bounds[0] = std::max(grid_bounds[0], 15.0f); grid_bounds[1] = std::min(grid_bounds[1], (float)vox_size[0] - 15.0f - 1.0f);
  grid_bounds[2] = std::max(grid_bounds[2], 15.0f); grid_bounds[3] = std::min(grid_bounds[3], (float)vox_size[1] - 15.0f - 1.0f);
  grid_bounds[4] = std::max(grid_bounds[4], 15.0f); grid_bounds[5] = std::min(grid_bounds[5], (float)vox_size[2] - 15.0f - 1.0f);
  // std::cout << grid_bounds[0] << " " << grid_bounds[1] << std::endl;
  // std::cout << grid_bounds[2] << " " << grid_bounds[3] << std::endl;
  // std::cout << grid_bounds[4] << " " << grid_bounds[5] << std::endl;
  int grid_size[3] = {0};
  grid_size[0] = grid_bounds[1] - grid_bounds[0] + 1;
  grid_size[1] = grid_bounds[3] - grid_bounds[2] + 1;
  grid_size[2] = grid_bounds[5] - grid_bounds[4] + 1;

  // Create list of hypothesis cubes (store grid locations, and is valid or not (0 for invalid, 1 for positive, 2 for negative))
  int num_hypothesis = grid_size[0] * grid_size[1] * grid_size[2];
  // std::cout << num_hypothesis << std::endl;
  unsigned short * hypothesis_locations = new unsigned short[3 * num_hypothesis];
  char * hypothesis_labels = new char[num_hypothesis];
  memset(hypothesis_labels, 0, sizeof(char) * num_hypothesis);
  for (int z = grid_bounds[4]; z <= grid_bounds[5]; z++)
    for (int y = grid_bounds[2]; y <= grid_bounds[3]; y++)
      for (int x = grid_bounds[0]; x <= grid_bounds[1]; x++) {
        int hypothesis_idx = (z - grid_bounds[4]) * grid_size[0] * grid_size[1] + (y - grid_bounds[2]) * grid_size[0] + (x - grid_bounds[0]);
        hypothesis_locations[0 * num_hypothesis + hypothesis_idx] = (unsigned short)x;
        hypothesis_locations[1 * num_hypothesis + hypothesis_idx] = (unsigned short)y;
        hypothesis_locations[2 * num_hypothesis + hypothesis_idx] = (unsigned short)z;
      }

  // Copy list of hypothesis cubes to GPU memory
  unsigned short * d_hypothesis_locations;
  char * d_hypothesis_labels;
  hipMalloc(&d_hypothesis_locations, 3 * num_hypothesis * sizeof(unsigned short));
  hipMalloc(&d_hypothesis_labels, num_hypothesis * sizeof(char));
  checkCUDA(__LINE__, hipGetLastError());
  hipMemcpy(d_hypothesis_locations, hypothesis_locations, 3 * num_hypothesis * sizeof(unsigned short), hipMemcpyHostToDevice);
  hipMemcpy(d_hypothesis_labels, hypothesis_labels, num_hypothesis * sizeof(char), hipMemcpyHostToDevice);
  checkCUDA(__LINE__, hipGetLastError());

  // Copy hypothesis crop information and object center to GPU memory
  unsigned short * d_hypothesis_crop_2D;
  float * d_object_center_cam;
  hipMalloc(&d_hypothesis_crop_2D, 4 * num_hypothesis * sizeof(unsigned short));
  hipMalloc(&d_object_center_cam, 3 * sizeof(float));
  checkCUDA(__LINE__, hipGetLastError());
  hipMemcpy(d_object_center_cam, object_center_cam, 3 * sizeof(float), hipMemcpyHostToDevice);
  checkCUDA(__LINE__, hipGetLastError());

  // Run kernel to get labels for hypotheses
  CUDA_NUM_THREADS = 512;
  CUDA_NUM_BLOCKS = (int)ceil(((float)num_hypothesis) / ((float)CUDA_NUM_THREADS));
  gen_hypothesis_labels <<< CUDA_NUM_BLOCKS, CUDA_NUM_THREADS >>>(num_hypothesis, d_hypothesis_locations, d_hypothesis_labels, d_hypothesis_crop_2D, d_object_center_cam, d_K, vox_unit, d_vox_size, d_vox_range_cam, d_vox_tsdf);

  // Copy 2D crop information back to CPU
  unsigned short * hypothesis_crop_2D = new unsigned short[4 * num_hypothesis];
  hipMemcpy(hypothesis_labels, d_hypothesis_labels, num_hypothesis * sizeof(char), hipMemcpyDeviceToHost);
  hipMemcpy(hypothesis_crop_2D, d_hypothesis_crop_2D, 4 * num_hypothesis * sizeof(unsigned short), hipMemcpyDeviceToHost);

  int num_invalid_hypotheses = 0;
  int num_positive_hypotheses = 0;
  int num_negative_hypotheses = 0;
  for (int i = 0; i < num_hypothesis; i++) {
    if (((int)hypothesis_labels[i]) == 0)
      num_invalid_hypotheses++;
    if (((int)hypothesis_labels[i]) == 1) {
      num_positive_hypotheses++;
      // std::cout << (int)hypothesis_locations[0 * num_hypothesis + i] << " " << (int)hypothesis_locations[1 * num_hypothesis + i] << " " << (int)hypothesis_locations[2 * num_hypothesis + i] << std::endl;
      // std::cout << (int)hypothesis_crop_2D[0 * num_hypothesis + i] << " " << (int)hypothesis_crop_2D[1 * num_hypothesis + i] << " " << (int)hypothesis_crop_2D[2 * num_hypothesis + i] << " " << (int)hypothesis_crop_2D[3 * num_hypothesis + i] << std::endl;
      // std::cout << std::endl;
      // cv::Rect curr_patch_ROI((int)hypothesis_crop_2D[0 * num_hypothesis + i], (int)hypothesis_crop_2D[1 * num_hypothesis + i], (int)hypothesis_crop_2D[2 * num_hypothesis + i], (int)hypothesis_crop_2D[3 * num_hypothesis + i]);
      // cv::Mat curr_patch = curr_frame_color(curr_patch_ROI);
      // cv::resize(curr_patch, curr_patch, cv::Size(227, 227));
      // cv::imshow("Patch", curr_patch);
      // cv::waitKey(0);
    }
    if (((int)hypothesis_labels[i]) == 2)
      num_negative_hypotheses++;
  }
  int num_valid_hypotheses = num_positive_hypotheses + num_negative_hypotheses;
  std::cout << "    Number of positive hypotheses found: " << num_positive_hypotheses << std::endl;
  std::cout << "    Number of negative hypotheses found: " << num_negative_hypotheses << std::endl;

  // Save to binary file: 8 x num_valid_hypotheses (int) (label, grid location (x,y,z), hypothesis 2D patch (x,y,width,height))
  std::string labels_filename = curr_sequence_directory + "/" + curr_frame_name + ".labels.bin";
  int * train_labels = new int[num_valid_hypotheses * 8 + 1];
  train_labels[0] = num_valid_hypotheses;
  int train_idx = 0;
  for (int i = 0; i < num_hypothesis; i++) {
    if (((int)hypothesis_labels[i]) > 0) {
      train_labels[0 * num_valid_hypotheses + train_idx + 1] = (int)hypothesis_labels[i];
      train_labels[1 * num_valid_hypotheses + train_idx + 1] = (int)hypothesis_locations[0 * num_hypothesis + i];
      train_labels[2 * num_valid_hypotheses + train_idx + 1] = (int)hypothesis_locations[1 * num_hypothesis + i];
      train_labels[3 * num_valid_hypotheses + train_idx + 1] = (int)hypothesis_locations[2 * num_hypothesis + i];
      train_labels[4 * num_valid_hypotheses + train_idx + 1] = (int)hypothesis_crop_2D[0 * num_hypothesis + i];
      train_labels[5 * num_valid_hypotheses + train_idx + 1] = (int)hypothesis_crop_2D[1 * num_hypothesis + i];
      train_labels[6 * num_valid_hypotheses + train_idx + 1] = (int)hypothesis_crop_2D[2 * num_hypothesis + i];
      train_labels[7 * num_valid_hypotheses + train_idx + 1] = (int)hypothesis_crop_2D[3 * num_hypothesis + i];
      train_idx++;
    }
  }
  // for (int i = 0; i < num_valid_hypotheses; i++) {
  //   for (int j = 0; j < 8; j++)
  //     std::cout << train_labels[j * num_valid_hypotheses + i + 1] << " ";
  //   std::cout << std::endl;
  // }
  std::ofstream tmp_out(labels_filename, std::ios::binary | std::ios::out);
  for (int i = 0; i < num_valid_hypotheses * 8 + 1; i++)
    tmp_out.write((char*)&train_labels[i], sizeof(int));
  tmp_out.close();










  return 0;
}

